#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "ImageUtil.h"
#include <iostream>
#include <fstream>
#include <string>
#include <sstream>
#include <iomanip>

//smooth out the potential to avoid divide by zero errors
#define EPS2 0.00001 

//GPU parameters
const int nThreads=1024;

/*Number of particles to skip in input. nSkip=1 
  gives 100,000 particles. nSkip=10 gives 10,000 
  particles. Overall 100,000/nSkip total particles. */
const int nSkip=10;

//Produce only nFrames worth of animation. 10 seconds at 30fps is 300 frames, which is a good number for homeworks.
const int nFrames=30;



// GPU Gems style HANDLE_ERROR function
static void HandleError(hipError_t err,
        const char *file,
        int line) {
    if (err != hipSuccess) {
        std::cout << hipGetErrorString(err) << " in " << file << " at line " << line << std::endl;
        exit(EXIT_FAILURE);
    }
}
#define HANDLE_ERROR( err ) (HandleError( err, __FILE__, __LINE__ ))


//Some string manipulation functions for saving files. pad_int(1234,5) returns "01234". 
std::string pad_int(int arg, int padcount) {
    std::stringstream ss;
    ss << std::setfill('0') << std::setw(padcount) << arg;
    return ss.str();
}

//Returns a file name in the form of "prefix00###suffix". For example "image0032.bmp"
std::string getFilename(std::string prefix, int num, int padcount, std::string suffix) {
    return prefix + pad_int(num, padcount) + suffix;
}


//Hybrid of GPU Gems 3 ch. 31 and CUDA nbody example.
//20 FLOPs
__device__ float3 bodyBodyInteraction(float4 bi, float4 bj, float3 ai) {
    float3 r;
    // r_ij  [3 FLOPs]
    r.x = bj.x - bi.x;
    r.y = bj.y - bi.y;
    r.z = bj.z - bi.z;
    // distSqr = dot(r_ij, r_ij) + EPS^2  [6 FLOPs]
    float distSqr = r.x * r.x + r.y * r.y + r.z * r.z + EPS2;
    // invDistCube =1/distSqr^(3/2)  [4 FLOPS (2 mul, 1 sqrt, 1 inv)]
    float distSixth = distSqr * distSqr * distSqr;
    float invDistCube = 1.0f / sqrtf(distSixth);
    // s = m_j * invDistCube [1 FLOP]
    float s = bj.w * invDistCube;
    // a_i =  a_i + s * r_ij [6 FLOPs]
    ai.x += r.x * s;
    ai.y += r.y * s;
    ai.z += r.z * s;
    return ai;
}

/* Hybrid of GPU Gems 3 ch. 31 and samples/5_Simulations/nbody/bodyststemcuda.cu */
__device__ float4 calculate_acceleration(float4 *devX, int numParticles, int deviceOffset) {
    __shared__ float4 shPosition[nThreads];
    float4 myPosition;
    int tile;
    float3 acc = { 0.0f, 0.0f, 0.0f };
    int gtid = deviceOffset + blockIdx.x * blockDim.x + threadIdx.x;
    myPosition = devX[gtid];
    for (tile = 0; tile<numParticles / blockDim.x; tile++) {
        shPosition[threadIdx.x] = devX[tile * blockDim.x + threadIdx.x];
        __syncthreads();
#pragma unroll 64
        for (unsigned int j = 0; j < blockDim.x; j++) {
            acc = bodyBodyInteraction(myPosition, shPosition[j], acc);
        }
        __syncthreads();
    }
    float4 acc4 = { acc.x, acc.y, acc.z, 0.0f };
    return acc4;
}

__global__ void kernel_step(float4 *devX, float4 *devV, float dt, int numParticles, int deviceOffset) {
    int index = deviceOffset + blockIdx.x * blockDim.x + threadIdx.x;

    //6 FLOPs
    devX[index].x += devV[index].x*dt;
    devX[index].y += devV[index].y*dt;
    devX[index].z += devV[index].z*dt;

    float4 acc = calculate_acceleration(devX, numParticles, deviceOffset);

    //6 FLOPs.
    devV[index].x += acc.x*dt;
    devV[index].y += acc.y*dt;
    devV[index].z += acc.z*dt;
}


int main()
{
    int nParticles;

    //Particle loading code! 
    std::vector<float4> particlepos;
    std::vector<float4> particlevel;
    //from http://stackoverflow.com/a/8421315/1030718
    std::ifstream source;                    // build a read-Stream
    source.open("galaxy1.txt", std::ios_base::in);  // open data
    int ctr=-1;
    for (std::string line; std::getline(source, line); )   //read stream line by line
    {
        ctr++;
        std::istringstream in(line);      //make a stream for the line itself
        if(ctr%nSkip!=0)
            continue;
        float m;
        float x;
        float y;
        float z;
        float vx;
        float vy;
        float vz;
        in >> m >> x >> y >> z >> vx >> vy >> vz;
        //multiplying my nSkip makes sure that if we only have one particle for every 10 particles in the data file, it has 10 times the mass.
        particlepos.push_back(make_float4(x,y,z,m*nSkip ));
        particlevel.push_back(make_float4( vx,vy,vz,0.0 ));
    }
    nParticles = (int)particlepos.size();
    //ensure nParticles is a multiple of nThreads*deviceCount
    int nParticlesNew = nThreads*int(float(nParticles) / (nThreads));
    if (nParticles != nParticlesNew) {
        std::cout << "WARNING in GPU_Phy::initialize. Position list argument size is not a multiple of nThreads*deviceCount!" << std::endl;
        std::cout << "Continuing by chopping to nearest multiple." << std::endl;
        nParticles = nParticlesNew; //this rounds down. nParticles<=positions.size().
    }

    float4 *host_x = new float4[nParticles];
    float4 *host_v = new float4[nParticles];
    for (int i = 0; i < nParticles; i++) {
        host_x[i] = particlepos[i];
        host_v[i] = particlevel[i];
    }

    int nBlocks = nParticles / (nThreads);

    HANDLE_ERROR(hipSetDevice(0));

    //initialize device memory
    float4 *dev_x;
    float4 *dev_v;

    //Allocate the memory on the device
    HANDLE_ERROR(hipMalloc((void**)(&dev_x), nParticles*sizeof(float4)));
    HANDLE_ERROR(hipMalloc((void**)(&dev_v), nParticles*sizeof(float4)));

    //Copy the memory to the device
    HANDLE_ERROR(hipMemcpy(dev_x, host_x, nParticles*sizeof(float4), hipMemcpyHostToDevice));
    HANDLE_ERROR(hipMemcpy(dev_v, host_v, nParticles*sizeof(float4), hipMemcpyHostToDevice));

    //timestep and save pictures
    float dt = 0.01;
    for (int i = 0; i < nFrames; i++) {
        std::cout << i << std::endl;
        //Do the computation
        for (int k = 0; k < 15; k++){
            kernel_step <<<nBlocks, nThreads >>>(dev_x, dev_v, dt, nParticles, 0);
            hipDeviceSynchronize();
        }

        //wait for the computation to finish
        hipDeviceSynchronize();

        //copy the memory to the computer
        HANDLE_ERROR(hipMemcpy(host_x, dev_x, nParticles*sizeof(float4), hipMemcpyDeviceToHost));

        //construct an image.
        //First fill up two arrays of scalars representing the particle density.
        //One for the dark matter (which will be reddish), and one for regular matter (bluish).
        DoubleImage regularMass(400, 400);
        DoubleImage darkMatter(400, 400);
        Image pic(400,400);
        for (int j = 0; j < nParticles; j++) {
            int x = int(host_x[j].x * 20) + 400 / 2;
            int y = int(host_x[j].y * 20) + 400 / 2;
            if(j<50000/nSkip){
                regularMass.increase(x, y, 1);
                regularMass.increase(x + 1, y, 0.5);
                regularMass.increase(x - 1, y, 0.5);
                regularMass.increase(x, y + 1, 0.5);
                regularMass.increase(x, y - 1, 0.5);
            } else {
                darkMatter.increase(x, y, 1);
                darkMatter.increase(x + 1, y, 0.5);
                darkMatter.increase(x - 1, y, 0.5);
                darkMatter.increase(x, y + 1, 0.5);
                darkMatter.increase(x, y - 1, 0.5);
            }
        }
        //Combine the two scalar arrays into an array of rgb values.
        for (int x = 0; x < 400; x++) {
            for (int y = 0; y < 400; y++) {
                //Human eye sees brightness logarithmically, so take a log.
                double scalar1 = log(1+3*regularMass.get(x, y))/4.0 ;
                double scalar2 = log(1 + darkMatter.get(x, y)) / 3.0;
                //combine everything into a nice color.
                pic.put(x, y, floatToRGB(scalar2*1.2+scalar1, 0+scalar1*1.5, scalar1*1.7));
            }
        }
        //actually save the image
        pic.save(getFilename("out/image", i, 3, ".bmp"));

    }


    delete [] host_x;
    delete [] host_v;
    HANDLE_ERROR(hipFree(dev_x));
    HANDLE_ERROR(hipFree(dev_v));
    hipDeviceReset();

    return 0;
}
